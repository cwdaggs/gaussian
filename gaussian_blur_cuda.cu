
#include <hip/hip_runtime.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define TILE_WIDTH 32
#define DIV_ROUND_UP(n, d)  (((n) + (d) - 1) / (d))
#define cuda_check(ret) _cuda_check((ret), __FILE__, __LINE__)
inline void _cuda_check(hipError_t ret, const char *file, int line) {
    if (ret != hipSuccess) {
        fprintf(stderr, "CudaErr: %s (%s:%d)\n", hipGetErrorString(ret), file, line);
        exit(1);    
    }
}

__constant__ int width_d;
__constant__ int height_d;
__constant__ int order_d;

__global__ void gaussian_calc_kernel(unsigned char *image_mat, unsigned char *result_mat, float *kernel) 
{ //Naive implementation
    float val = 0;
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int j = blockIdx.x * blockDim.x + tx; //col
    int i = blockIdx.y * blockDim.y + ty; //row

    if (i >= height_d || j >= width_d) {
        return;
    }

    int center = (order_d - 1) / 2;

    for (int x = 0; x < order_d; x++) {
        for (int y = 0; y < order_d; y++) {
            int mat_x = max(0, min(height_d - 1, i + x - center));
            int mat_y = max(0, min(width_d - 1, j + y - center));

            val += image_mat[mat_x * height_d + mat_y] * kernel[x * order_d + y];
            // if (x <= center - i && y <= center - j) {
            //     val += (image_mat[0] * kernel[x * order_d + y]);
            // } else if (x <= center - i && j + y >= width_d + center - 1) {
            //     val += (image_mat[width_d - 1] * kernel[x * order_d + y]);
            // } else if (i + x >= height_d + center - 1 && y <= center - j) {
            //     val += (image_mat[height_d * (height_d - 1)] * kernel[x * order_d + y]);
            // } else if (i + x >= height_d + center - 1 && j + y >= width_d + center - 1) {
            //     val += (image_mat[height_d * (height_d - 1) + width_d - 1] * kernel[x * order_d + y]);
            // } else if (x < center - i) {
            //     val += (image_mat[j + (y - center)] * kernel[x * order_d + y]);
            // } else if (y < center - j) {
            //     val += (image_mat[(i + (x - center)) * height_d] * kernel[x * order_d + y]);
            // } else if (i + x > height_d + center - 1) {
            //     val += (image_mat[height_d * (height_d - 1) + j + (y - center)] * kernel[x * order_d + y]);
            // } else if (j + y > width_d + center - 1) {
            //     val += (image_mat[(i + (x - center)) * height_d + width_d - 1] * kernel[x * order_d + y]);
            // } else {
            //     val += (image_mat[(i + (x - center)) * height_d + j + (y - center)] * kernel[x * order_d + y]);
            // }
        }
    }
    result_mat[i * height_d + j] = (unsigned char) val; 
    val = 0;
}


// __global__ void gaussian_calc_kernel(unsigned char *image_mat, unsigned char *result_mat, float *kernel, int width, int height, float order) 
// { // Shared memory implementation
//     __shared__ unsigned char image_s[TILE_WIDTH][TILE_WIDTH];
//     int tx = threadIdx.x;
//     int ty = threadIdx.y;
//     int j = blockIdx.x * TILE_WIDTH + tx; //col
//     int i = blockIdx.y * TILE_WIDTH + ty; //row

//     if (j >= height || i >= width) {
//         return;
//     }
//     image_s[ty][tx] = image_mat[i * width + j];
//     __syncthreads();
//     float val = 0;
    

//     int center = ((int) order - 1) / 2;

//     for (int x = 0; x < (int) order; x++) {
//         for (int y = 0; y < (int) order; y++) {
//             if (x <= center - i && y <= center - j) {
//                 val += (image_s[0][0] * kernel[x * (int) order + y]);
//             } else if (x <= center - i && j + y >= width + center - 1) {
//                 val += (image_s[0][TILE_WIDTH - 1] * kernel[x * (int) order + y]);
//             } else if (i + x >= height + center - 1 && y <= center - j) {
//                 val += (image_s[TILE_WIDTH - 1][0] * kernel[x * (int) order + y]);
//             } else if (i + x >= height + center - 1 && j + y >= width + center - 1) {
//                 val += (image_s[TILE_WIDTH - 1][TILE_WIDTH - 1] * kernel[x * (int) order + y]);
//             } else if (x < center - i) { //row too high
//                 val += (image_s[0][j + (y - center)] * kernel[x * (int) order + y]);
//             } else if (y < center - j) { // col too far left
//                 val += (image_s[(i + (x - center)) * TILE_WIDTH][0] * kernel[x * (int) order + y]);
//             } else if (i + x > height + center - 1) { // row too low
//                 val += (image_s[TILE_WIDTH - 1][j + (y - center)] * kernel[x * (int) order + y]);
//             } else if (j + y > width + center - 1) { // col too far right
//                 val += (image_s[(i + (x - center)) * height + width - 1][TILE_WIDTH - 1] * kernel[x * (int) order + y]);
//             } else {
//                 val += (image_s[i][j] * kernel[x * (int) order + y]);
//             }
//         }
//     }
//     result_mat[i * height + j] = (unsigned char) val; 
//     val = 0;
// }

void gaussian_calc(unsigned char *image_mat, unsigned char *result_mat, float *kernel, int width, int height, int order) 
{
    /* Allocate device memory for all matrices */
    float *kernel_d;
    unsigned char *image_mat_d, *result_mat_d;
    cuda_check(hipMalloc(&kernel_d, order * order * sizeof(float)));
    cuda_check(hipMalloc(&image_mat_d, width * height * sizeof(unsigned char)));
    cuda_check(hipMalloc(&result_mat_d, width * height * sizeof(unsigned char)));
    
    /* Copy kernel and image_mat to device */
    cuda_check(hipMemcpy(kernel_d, kernel, order * order * sizeof(float), hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(image_mat_d, image_mat, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));
    cuda_check(hipMemcpyToSymbol(HIP_SYMBOL(width_d), &width, sizeof(int)));
    cuda_check(hipMemcpyToSymbol(HIP_SYMBOL(height_d), &height, sizeof(int)));
    cuda_check(hipMemcpyToSymbol(HIP_SYMBOL(order_d), &order, sizeof(int)));

    /* Invoke kernel function */
    dim3 block_dim(32, 32);
    dim3 grid_dim(DIV_ROUND_UP(height, block_dim.x), DIV_ROUND_UP(width, block_dim.y));
    gaussian_calc_kernel<<<grid_dim, block_dim>>>(image_mat_d, result_mat_d, kernel_d);

    /* Copy result back to host */
    cuda_check(hipMemcpy(result_mat, result_mat_d, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));

    /* Free device memory */
    cuda_check(hipFree(kernel_d));
    cuda_check(hipFree(image_mat_d));
    cuda_check(hipFree(result_mat_d));
}

void write_gaussian(char *filename, unsigned char *picture, int width, int height)
{
	FILE *fp;

	/* Open file */
	fp = fopen(filename, "wb");
	if (!fp) {
		fprintf(stderr, "Error: cannot open file %s", filename);
		exit(1);
	}

	/* Put structural information */
	fprintf(fp, "P5\n%d %d\n255\n", width, height);

	/* Output grayscale pixels */
	fwrite(picture, sizeof(unsigned char), width * height, fp);

	// free(pixels);
	fclose(fp);
}

int main(int argc, char *argv[])
{
	float sigma, order;
	char *output_filename;
    int width, height;
    FILE *input_file;

	/* Command line arguments */
	if (argc < 4) {
		fprintf(stderr, "Usage: %s <input_pgm> <output_pgm> <sigma>\n",
				argv[0]);
		exit(1);
	}

    input_file = fopen(argv[1], "rb");
    if (!input_file) {
        fprintf(stderr, "Error: cannot open file %s", argv[1]);
		exit(1);
    }
    output_filename = argv[2];

    if (fscanf(input_file, "%*[^\n]\n") != 0) {
        exit(1);
    }
    if (fscanf(input_file, "%d %d\n", &width, &height) != 2) {
        exit(1);
    }
    if (fscanf(input_file, "%*[^\n]\n") != 0) {
        exit(1);
    }
    
    sigma = atoi(argv[3]);
    if (sigma <= 0) {
        fprintf(stderr, "Error: invalid sigma value");
		exit(1);
    }
    order = ceil(6 * sigma);
    if ((int)order % 2 == 0) {
        order++;
    }
    if (order > width || order > height) {
        fprintf(stderr, "Error: sigma value too big for image size");
		exit(1);
    }

    float *kernel;
    unsigned char *image_mat, *result_mat;
    kernel = (float*)aligned_alloc(64, (int) order * (int) order * sizeof(float));
    image_mat = (unsigned char*)aligned_alloc(64, width * height * sizeof(unsigned char));
    result_mat = (unsigned char*)aligned_alloc(64, width * height * sizeof(unsigned char));

    if (fread(image_mat, sizeof(unsigned char), height * width, input_file) != (size_t)(height * width)) {
        exit(1);
    }
    fclose(input_file);
    
    float sum = 0;
    // for (int i = 0; i < order; i++) {
    //     for (int j = 0; j < order; j++) {
    //         kernel[i * (int) order + j] = (1/(2*M_PI*sigma*sigma)) * 
    //         exp(-(pow(i - floor(order/2), 2) + pow(j - floor(order/2), 2))/(2 * sigma * sigma));
    //         // printf("%.8f ", kernel[i * (int) order + j]);
    //     }
    //     // printf("\n");
    // }

    for (int i = 0; i < order; i++) {
        for (int j = 0; j < order; j++) {
            kernel[i * (int) order + j] = exp(-(pow(i - floor(order/2), 2) + pow(j - floor(order/2), 2))/(2 * sigma * sigma));
            sum += exp(-(pow(i - floor(order/2), 2) + pow(j - floor(order/2), 2))/(2 * sigma * sigma));
            // printf("%.8f ", kernel[i * (int) order + j]);
        }
        // printf("\n");
    }

    for (int i = 0; i < order; i++) {
        for (int j = 0; j < order; j++) {
            kernel[i * (int) order + j] /= sum;
            // printf("%.8f ", kernel[i * (int) order + j]);
        }
        // printf("\n");
    }
    
    gaussian_calc(image_mat, result_mat, kernel, width, height,(int) order);

    // /* Save output image */
	write_gaussian(output_filename, result_mat, width, height);
    
	// free(filename);

	return 0;
}